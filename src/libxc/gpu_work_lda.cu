#include "hip/hip_runtime.h"

typedef void (*lda_ptr)(const void *p,  xc_lda_work_t *r);

#include "gpu_fsign_lda.h"

__device__ void gpu_work_lda_c(gpu_libxc_info* glinfo, const double d_rhoa, const double d_rhob, double *d_zk, double *d_vrho, int nspin){

#ifdef DEBUG 
		 //printf("FILE: %s, LINE: %d, FUNCTION: %s, GPU_WORK_LDA.. \n", __FILE__, __LINE__, __func__);
		//printf("func_id: %d rho: %f \n", glinfo->func_id, (d_rhoa+d_rhob));
#endif	

	gpu_libxc_info* d_glinfo;
	d_glinfo = (gpu_libxc_info*)glinfo;

	gpu_lda_work_params *d_w;
	d_w = (gpu_lda_work_params*)(d_glinfo->d_worker_params);

	xc_lda_work_t r;
	r.order = 1;
	r.nspin = nspin;

	double dens, drs;
	gpu_xc_rho2dzeta(nspin, d_rhoa, d_rhob, &dens, &(r.z));

#ifdef DEBUG 
		//printf("func_id: %d rho: %f dens: %f r.z: %f \n", glinfo->func_id, (d_rhoa+d_rhob), dens, r.z);
#endif

	if(dens > d_w->dens_threshold){

	r.rs = d_w->cnst_rs*pow(dens, -1.0/d_w->xc_dim);

#ifdef DEBUG 
                //printf("func_id: %d rho: %f dens: %f r.z: %f r.rs: %f d_w->cnst_rs: %f d_w->xc_dim: %d \n", glinfo->func_id, (d_rhoa+d_rhob), dens, r.z, r.rs, d_w->cnst_rs, d_w->xc_dim);
#endif


#ifdef DEBUG 
                 //printf("FILE: %s, LINE: %d, FUNCTION: %s, CALLING MAPLE2C FUNCTION \n", __FILE__, __LINE__, __func__);
#endif

	(maple2cf_lda[d_w->k_index])(d_glinfo->d_maple2c_params, &r);

	*d_zk = r.f;

	if(r.order > 0){
		drs = -r.rs/(d_w->xc_dim*dens);

		*d_vrho = r.f + dens*r.dfdrs*drs;
		/*if(p->nspin == XC_POLARIZED){
			vrho[1] = vrho[0] - (r.z + 1.0)*r.dfdz;
			vrho[0] = vrho[0] - (r.z - 1.0)*r.dfdz;
		}*/
	}
	

	}
}
